#include <iostream> 
#include <hip/hip_runtime.h>

__global__ 
void vecAdd(float *a, float*b, float*c, int n){
    int block_id = blockIdx.x;
    int block_width = blockDim.x;
    int thread_id = threadIdx.x;
    int index = block_id * block_width + thread_id;
    if (index < n){
        c[index] = a[index] + b[index];
    }

}


int main(){
    const int n = 1024;
    const int size = n*sizeof(float);
    
    // host mem allocation
    float* A_h= new float[n];
    float* B_h= new float[n];
    float* C_h= new float[n];

    for (int i=0; i<n; ++i){
        A_h[i] = 1.0f;
        B_h[i] = 2.0f;
    }

    // device mem allocation
    float *A_d, *B_d, *C_d;
    hipMalloc((void **)&A_d, size);
    hipMalloc((void **)&B_d, size);
    hipMalloc((void **)&C_d, size);

    hipMemcpy(A_d, A_h,size,hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h,size,hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock-1)/threadsPerBlock;

    vecAdd<<<blocksPerGrid,threadsPerBlock>>>(A_d,B_d,C_d,n);

    hipMemcpy(C_h,C_d,size,hipMemcpyDeviceToHost);
    
    std::cout << "Checking Error in computation..."<<std::endl;
        for (int i=0; i<n; i++){
        float val = 3.0f;
        std::cout << "Error at index: " << i << ": " << C_h[i] - val << std::endl;
    }
    
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
    delete[] A_h;
    delete[] B_h;
    delete[] C_h;
    
    return 0;
}